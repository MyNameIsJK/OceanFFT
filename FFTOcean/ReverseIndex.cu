#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_vector_types.h>

// binaryLen��ʾҪת���Ķ����Ƴ���
__device__ int reverseDev(uint index, uint binaryLen)
{
	uint ret = 0;
	for (int i = 0; i < binaryLen; i++)
	{
		ret = (ret * 2) + (index & 1);
		index /= 2;
	}
	return ret;
}

__global__ void reverseIndexKernel(uint* reverseIndex, uint arrayLen, uint binaryLen)
{
	uint idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < arrayLen)
	{
		reverseIndex[idx] = reverseDev(idx, binaryLen);
	}
}

extern "C" void generateReverseIndex(uint * reverseIndex, uint arrayLen, uint binaryLen, dim3 blockSize, dim3 gridSize)
{
	reverseIndexKernel << <gridSize, blockSize >> > (reverseIndex, arrayLen, binaryLen);
}